
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include<iostream>

using namespace std;

#define WARP_SIZE 32
#define WARP_COUNT 16
#define BLOCK_COUNT 13

double *u_CPU, *v_CPU;
double *u_GPU, *v_GPU;

void allocateAndInitializeInputVectors(int vectorLength);
void die(const char *error); 
void check_error(hipError_t e);
 
__global__ void computePairwiseMultInGPU(double *u, const double *v, int vectorLength);
__global__ void computeSumInGPU(double *u, int vectorLength);
__device__ void computeWarpSum(double *elements, int threadId);

int main(int argc, char **argv) {

	int vectorLength = (argc > 1) ? atoi(argv[1]) : 100000;

	clock_t start = clock();	
	allocateAndInitializeInputVectors(vectorLength);
	clock_t end = clock();
        double elapsed = (end - start) / (double) CLOCKS_PER_SEC;
        cout << "Allocation time: " << elapsed << " seconds \n";
	start = clock();
	int threadsPerBlock = WARP_SIZE * WARP_COUNT;
	computePairwiseMultInGPU<<< BLOCK_COUNT, threadsPerBlock >>>(u_GPU, v_GPU, vectorLength);
	computeSumInGPU<<< 1, threadsPerBlock >>>(u_GPU, vectorLength);
	hipDeviceSynchronize();

	check_error(hipGetLastError());
	double dotProduct = 0;
	check_error(hipMemcpy(&dotProduct, u_GPU, sizeof(double), hipMemcpyDeviceToHost));
	end = clock();
	elapsed = (end - start) / (double) CLOCKS_PER_SEC;
	cout << "Dot product of the vectors: " << dotProduct << "\n";
	cout << "Execution time: " << elapsed << " seconds \n";

	return 0;
}

void allocateAndInitializeInputVectors(int vectorLength) {

	size_t vectorSize = vectorLength * sizeof(double);
	u_CPU = (double*) malloc(vectorSize);
	check_error(hipMalloc((void **) &u_GPU, vectorSize));
	srand(time(NULL));
  	for (int i = 0; i < vectorLength; i++) {
		u_CPU[i] = 6 / (1.00 + (rand() % 5)); 
	}
	check_error(hipMemcpyAsync(u_GPU, u_CPU, vectorSize, hipMemcpyHostToDevice, 0));
	v_CPU = (double*) malloc(vectorSize);
	check_error(hipMalloc((void **) &v_GPU, vectorSize));
  	for (int i = 0; i < vectorLength; i++) {
		v_CPU[i] = 10 / (1.00 + rand() % 10); 
	}
	check_error(hipMemcpyAsync(v_GPU, v_CPU, vectorSize, hipMemcpyHostToDevice, 0));
}
void die(const char *error) {
        printf("%s", error);
        exit(1);
}

void check_error(hipError_t e) {
        if (e != hipSuccess) {
                printf("\nCUDA error: %s\n", hipGetErrorString(e));
                exit(1);
        }
}

__global__ void computePairwiseMultInGPU(double *u, const double *v, int vectorLength) {
	int startIndex = blockIdx.x + threadIdx.x;
	int stride = BLOCK_COUNT * WARP_COUNT * WARP_SIZE;
	for (int i = startIndex; i < vectorLength; i += stride) {
		u[i] = u[i] * v[i];
	}

}
__global__ void computeSumInGPU(double *u, int vectorLength) {
	__shared__ double elements_to_sum[WARP_COUNT][WARP_SIZE];
	__shared__ double partial_sums[WARP_COUNT];
	int threadId = threadIdx.x % WARP_SIZE;
        int warpId = threadIdx.x / WARP_SIZE;
	for (int i = 0; i < WARP_COUNT; i++) {
		if (threadId == 0) {
			partial_sums[warpId] = 0.0;
		}
	}
	for (int i = WARP_SIZE * warpId; i < vectorLength; i += WARP_SIZE * WARP_COUNT) {
		int startIndex = i;
		int endIndex = (vectorLength < i + WARP_SIZE) 
			? startIndex + WARP_SIZE - 1 : vectorLength - 1;
		if (startIndex + threadId <= endIndex) {
			elements_to_sum[warpId][threadId] = u[startIndex + threadId];
		} else {
			elements_to_sum[warpId][threadId] = 0;
		}
		computeWarpSum(elements_to_sum[warpId], threadId);
		if (threadId == 0) {
			partial_sums[warpId] += elements_to_sum[warpId][0];
		}
	}
	__syncthreads();
	if (warpId == 0 && threadId == 0) {
		for (int i = 1; i < WARP_COUNT; i++) {
			partial_sums[0] += partial_sums[i];
		}
		u[0] = partial_sums[0];
	}
}

__device__ void computeWarpSum(double *elements, int threadId) {

	if (threadId < 16) elements[threadId * 2] = elements[threadId * 2] + elements[threadId * 2 + 1];
	if (threadId < 8) elements[threadId * 4] = elements[threadId * 4] + elements[threadId * 4 + 2];
	if (threadId < 4) elements[threadId * 8] = elements[threadId * 8] + elements[threadId * 8 + 4];
	if (threadId < 2) elements[threadId * 16] = elements[threadId * 16] + elements[threadId * 16 + 8];
	if (threadId == 0) elements[0] = elements[0] + elements[16];
}
